
#include <iostream>
#include <hip/hip_runtime.h>


// called only from device 
__device__ void costFunction(float* F, float* T, int i, int numElements){

	if (i < numElements)
	{
		F[i] = T[i] * T[i];
	}
	
}

// Trying finite difference, called from host
__global__ void finiteDiff(float* Fp, float* Tp, float* Fn, float* Tn, float* dF, float e, int numElements){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	costFunction(Fn,Tn,i,numElements);
	costFunction(Fp,Tp,i,numElements);
	if (i < numElements)
	{
		dF[i] = (Fp[i] - Fn[i]) / (2*e); // think the 0.0f exists to cast to float
	}
	
}




int main (void) {
    
    // Declare Vairables host variables start with h, device vars with d
	int numElements = 50000; // vector size --- determine whats the max we can do on GPU
	size_t size = numElements*sizeof(float); // total vector size

	// we want C <-- A*B so init three vectors on host
	float *h_Fp = (float*) malloc(size);
	float *h_Fn = (float*) malloc(size);
	float *h_Tp = (float*) malloc(size);
    float *h_Tn = (float*) malloc(size);
    float *h_dF = (float*) malloc(size);

	if (h_Fp == NULL || h_Fn == NULL || h_Tp == NULL || h_Tn == NULL)
	{
		std::cout<< "Failed to allocate on host device" << std::endl;
		exit(EXIT_FAILURE);
	}

    // initialize Tp and Tn
    float e = 0.0001;
    for (int  i = 0; i < numElements; i++)
    {
        // 0.001 -> 500
        h_Tn[i] = i/100 - e;
        h_Tp[i] = i/100 + e;     
    }
    
    // DEVICE -- GPU

	hipError_t err = hipSuccess; // errror code 

	// init vectors on device
	float *d_Fp = NULL; // NULL pointer
	err = hipMalloc((void **)&d_Fp, size);
	if (err != hipSuccess)
	{
		std::cout << "Failed to allocate device vector d_Fp, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    float *d_Fn = NULL; // NULL pointer
	err = hipMalloc((void **)&d_Fn, size);
	if (err != hipSuccess)
	{
		std::cout << "Failed to allocate device vector d_Fn, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    float *d_Tp = NULL; // NULL pointer
	err = hipMalloc((void **)&d_Tp, size);
	if (err != hipSuccess)
	{
		std::cout << "Failed to allocate device vector d_Tp, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    float *d_Tn = NULL; // NULL pointer
	err = hipMalloc((void **)&d_Tn, size);
	if (err != hipSuccess)
	{
		std::cout << "Failed to allocate device vector d_Tn, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    float *d_dF = NULL; // NULL pointer
	err = hipMalloc((void **)&d_dF, size);
	if (err != hipSuccess)
	{
		std::cout << "Failed to allocate device vector d_dF, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    // COPY DATA FROM HOST TO DEVICE using cudaMemcpy(destination_add, source_add, size, direction)
	std::cout << "Begin to copy memory from host to device." << std::endl;

    err = hipMemcpy(d_Tp, h_Tp, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cout << "Failed to copy host vector Tp to device, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    err = hipMemcpy(d_Tn, h_Tn, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cout << "Failed to copy host vector Tp to device, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    // FINITE DIFF ON KERNEL
	int threadsPerBlock = 256; // threads per block is limited to 1024
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock; // dont understand this tbh
	std::cout<<"CUDA kernel lauched with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

    finiteDiff<<<blocksPerGrid,threadsPerBlock>>>(d_Fp,d_Tp,d_Fn,d_Tn, d_dF, e, numElements);
    err = hipGetLastError();
    if (err != hipSuccess)
	{
		std::cout << "Failed to lauch vecAdd kernel, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    // COPY DATA BACK TO HOST FROM DEVICE 
	std::cout << "Begin to copy memory from device to host." << std::endl;
	err = hipMemcpy(h_dF, d_dF, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		std::cout << "Failed to copy device vector dF to host, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}

    // for (int i = 0; i < 200; i++)
	// {
	// 	if (fabs(2*(h_Tn[i] + e) - h_dF[i]) > 1e-2)
	// 	{		
	// 		std::cout<<"Result verification failed at element " << i << " tol:: " << fabs(2*(h_Tn[i] + e) - h_dF[i]) << std::endl;
	// 		exit(EXIT_FAILURE);
	// 	}
	// }

    // free vectors on device
	err = hipFree(d_Fp);
	if (err != hipSuccess)
	{
		std::cout << "Failed to free device vector d_Fp, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
    err = hipFree(d_Fn);
	if (err != hipSuccess)
	{
		std::cout << "Failed to free device vector d_Fn, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
    err = hipFree(d_Tp);
	if (err != hipSuccess)
	{
		std::cout << "Failed to free device vector d_Tp, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
    err = hipFree(d_Tn);
	if (err != hipSuccess)
	{
		std::cout << "Failed to free device vector d_Tn, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}
    err = hipFree(d_dF);
	if (err != hipSuccess)
	{
		std::cout << "Failed to free device vector d_dF, Error code: " << hipGetErrorString(err) << std::endl;
		exit(EXIT_FAILURE);
	}




    // free memory
    free(h_Fp);
    free(h_Fn);
    free(h_Tp);
    free(h_Tn);
    free(h_dF);

    std::cout << "Done!" << std::endl;
    return 0;
}